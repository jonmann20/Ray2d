#include "hip/hip_runtime.h"
// OpenGL, OpenGl Utitliy Toolkit (GLUT)
#include <GL/glew.h>
#include <GL/freeglut.h>

// CUDA
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Box2d
#include <Box2D/Box2D.h>

// FlappyRay helpers
#include "utils.h"			// includes: iostream, namespace std
#include "globals.h"		// includes: vec2, player, light
#include "rect.h"			// includes: vec2
#include "circle.h"			// includes: vec2
#include "line.h"			// includes: vec2
#include "input.h"

// OpenGL globals
GLvoid* font_style = GLUT_BITMAP_HELVETICA_12;

struct CollisionResponse {
	Vec2 overlapN, overlapV;
};

#pragma region CUDA
__global__
void square(int* a) {
	*a = (*a) * (*a);
}


void testCuda() {
	int a = 8;
	int* da;	// device copy
	int size = sizeof(int);

	// Init da on GPU
	hipMalloc((void**)&da, size);
	hipMemcpy(da, &a, size, hipMemcpyHostToDevice);

	square << <1, 1 >> >(da);

	// Grab answer from GPU
	hipMemcpy(&a, da, size, hipMemcpyDeviceToHost);
	hipFree(da);

	cout << "a: " << a << endl;
}
#pragma endregion CUDA

#pragma region Update

CollisionResponse testLineRect(Line a, Rect b) {
	CollisionResponse r;
	r.overlapV = Vec2(0, 0);

	float x2 = b.pos.x + b.size.x;
	float y2 = b.pos.y + b.size.y;

	if(x2 < a.start.x) {			// player is not intersecting line.
		r.overlapV = Vec2(0, 0);
	}

	if(b.pos.x > a.end.x) {			// player is not intersecting line.
		r.overlapV = Vec2(0, 0);
	}
	
	//cout << y2 << " < " << a.start.y << " && " << b.pos.y << " > " << a.start.y << endl;

	if(y2 < a.start.y && b.pos.y > a.start.y) {		// bottom of player is below line 0, and player is intersecting line
		r.overlapV = Vec2(0, a.start.y - y2);
	}

	return r;
}

void checkRayCollision() {
	for(const auto& light : lights) {
		for(auto& chunk : player.body) {
		//auto chunk = player.body.back();
		
			float x = player.pos.x + chunk.pos.x;
			float y = player.pos.y + chunk.pos.y;
		
			//cout << x << " <= " << light.pos.x << " && " << (x + chunk.size.x) << " >= " << light.pos.x << endl;

			CollisionResponse response = testLineRect(light.rays.back(), Rect(x, y, chunk.size.x, chunk.size.y, chunk.color));

			if(response.overlapV.y != 0) {
				chunk.color = Vec3(1, 1, 1);
			}
			else {
				chunk.color = chunk.INIT_COLOR;
			}

			//if((x <= light.pos.x) &&
			//   ((x + chunk.size.x) >= light.pos.x)
			//){
			//	//cout << "new color" << endl;
			//	chunk.color = Vec3(1, 1, 1);
			//}
			//else {
			//	//cout << "init color" << endl;
			//	chunk.color = chunk.INIT_COLOR;
			//}
		}
	}
}

void update() {
	player.updatePos();
	checkRayCollision();

	glutPostRedisplay();
}

void calculateFPS() {
	++frameCount;

	// Get the number of milliseconds since glutInit called 
	// (or first call to glutGet(GLUT ELAPSED TIME)).
	currentTime = glutGet(GLUT_ELAPSED_TIME);

	// Calculate time passed
	int timeInterval = currentTime - previousTime;

	if(timeInterval > 1000) {
		// calculate the number of frames per second
		fps = frameCount / (timeInterval / 1000.0f);

		// Set time
		previousTime = currentTime;

		// Reset frame count
		frameCount = 0;
	}
}
#pragma endregion Update

#pragma region Render
void drawText(Vec2 pos, char* format, ...) {
	// Initialize a variable argument list
	va_list args;
	va_start(args, format);

	// Return the number of characters in the string referenced the list of arguments.
	// _vscprintf doesn't count terminating '\0' (that's why +1)
	int len = _vscprintf(format, args) + 1;

	// Allocate memory for a string of the specified size
	char* text = (char*)malloc(len * sizeof(char));

	// Write formatted output using a pointer to the list of arguments
	vsprintf_s(text, len, format, args);

	// End using variable argument list 
	va_end(args);

	// Specify the raster position for pixel operations
	glRasterPos2f(pos.x, pos.y);

	for(int i=0; text[i] != '\0'; ++i) {
		glutBitmapCharacter(font_style, text[i]);
	}

	free(text);
}

void drawFPS() {
	//  Load the identity matrix so that FPS string being drawn won't get animates
	glLoadIdentity();

	glColor3f(0.6, 0.6, 0);
	drawText(Vec2(DEBUG_INFOX, 0.92), "FPS: %4.2f", fps);
}

void drawLights() {
	for(auto light : lights) {
		light.draw();
	}
}

void render() {
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// Level
	drawLights();

	// Player
	player.draw();

	// Debug
	calculateFPS();
	drawFPS();

	//if(debugRays) {
	//	drawRays();
	//}
	//else {
	//	glColor3f(0.6, 0.6, 0);
	//	drawText(Vec2(DEBUG_INFOX, 0.87), "DebugRays Off");
	//}

	glutSwapBuffers();
}
#pragma endregion Render


int main(int argc, char* argv[]) {
	//----- Game Setup
	player = Player(0, 0, 0.2, 0.2);

	Vec3 warmFlourescent = Vec3(1, 0.95686, 0.89804);		// http://planetpixelemporium.com/tutorialpages/light.html
	lights.push_back(Light(0, 0.85, LightType::FLOURESCENT, warmFlourescent, true));


	//----- Box2D setup
	b2Vec2 gravity(0.f, -10.f);
	//b2World world(gravity);


	//----- OpenGL setup
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(720, 720);		// 1280 x 720
	glutInitWindowPosition(320, 180);
	glutCreateWindow("FlappyRay Engine Demo");

	//glutGameModeString("1280x720:16@60");		// 16 bits per pixel
	//glutEnterGameMode();

	glutDisplayFunc(render);
	glutIdleFunc(update);
	//glutTimerFunc(32, update, -1);

	glutIgnoreKeyRepeat(1);
	glutKeyboardFunc(keydown);
	glutKeyboardUpFunc(keyup);
	//glutSpecialFunc(keyboard);

	glutMainLoop();


	return EXIT_SUCCESS;
}